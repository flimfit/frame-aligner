#include "hip/hip_runtime.h"
#include "GpuFrameWarperKernels.h"
#include "GpuTextureManager.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"

const int num_tex = 16;
texture<float, 3, hipReadModeElementType> tex0;
texture<float, 3, hipReadModeElementType> tex1;
texture<float, 3, hipReadModeElementType> tex2;
texture<float, 3, hipReadModeElementType> tex3;
texture<float, 3, hipReadModeElementType> tex4;
texture<float, 3, hipReadModeElementType> tex5;
texture<float, 3, hipReadModeElementType> tex6;
texture<float, 3, hipReadModeElementType> tex7;
texture<float, 3, hipReadModeElementType> tex8;
texture<float, 3, hipReadModeElementType> tex9;
texture<float, 3, hipReadModeElementType> tex10;
texture<float, 3, hipReadModeElementType> tex11;
texture<float, 3, hipReadModeElementType> tex12;
texture<float, 3, hipReadModeElementType> tex13;
texture<float, 3, hipReadModeElementType> tex14;
texture<float, 3, hipReadModeElementType> tex15;

float3 operator+(const float3& a, const float3& b)
{
   float3 c;
   c.x = a.x + b.x;
   c.y = a.y + b.y;
   c.z = a.z + b.z;
   return c;
}

__device__ float3& operator+=(float3& a, const float3& b)
{
   a.x += b.x;
   a.y += b.y;
   a.z += b.z;
   return a;
}

__device__ float3 operator-(const float3& a, const float3& b)
{
   float3 c;
   c.x = a.x - b.x;
   c.y = a.y - b.y;
   c.z = a.z - b.z;
   return c;
}

__device__ float3& operator-=(float3& a, const float3& b)
{
   a.x -= b.x;
   a.y -= b.y;
   a.z -= b.z;
   return a;
}

class f3
{
public:

   __device__ f3& operator=(float other)
   {
      x = other;
      y = other;
      z = other;
      return *this;
   }

   __device__ f3& operator=(f3& other)
   {
      x = other.x;
      y = other.y;
      z = other.z;
      return *this;
   }

   __device__ f3& operator+=(const f3& other)
   {
      x += other.x;
      y += other.y;
      z += other.z;
      return *this;
   }

   float x = 0;
   float y = 0;
   float z = 0;
};

GpuTextureManager* GpuTextureManager::gpu_texture_manager = nullptr;

texture<float, 3, hipReadModeElementType>& getTexture(int id)
{
   switch (id)
   {
   case 0: return tex0;
   case 1: return tex1;
   case 2: return tex2;
   case 3: return tex3;
   case 4: return tex4;
   case 5: return tex5;
   case 6: return tex6;
   case 7: return tex7;
   case 8: return tex8;
   case 9: return tex9;
   case 10: return tex10;
   case 11: return tex11;
   case 12: return tex12;
   case 13: return tex13;
   case 14: return tex14;
   case 15: return tex15;
   }
   throw std::runtime_error("Invalid texture reference");
}

GpuTextureManager* GpuTextureManager::instance()
{
   if (!gpu_texture_manager)
      gpu_texture_manager = new GpuTextureManager;
   return gpu_texture_manager;
}

int GpuTextureManager::getTextureId()
{
   std::unique_lock<std::mutex> lk(mutex);
   cv.wait(lk, [this] { return !free_textures.empty(); });
   int texture = free_textures.back();

   free_textures.pop_back();
   return texture;
}

void GpuTextureManager::returnTextureId(int t)
{
   {
      std::lock_guard<std::mutex> lk(mutex);
      free_textures.push_back(t);
   }
   cv.notify_one();
}

GpuTextureManager::GpuTextureManager()
{
   for (int i = 0; i < num_tex; i++)
      free_textures.push_back(i);
}


template <unsigned int blockSize, class T>
__device__ void warpReduce(T *sdata, unsigned int tid) {

}

template <unsigned int blockSize, class T>
__global__ void reduceSum(T *g_idata, T *g_odata, unsigned int n) {
   extern __shared__ char sdata_[];
   T* sdata = reinterpret_cast<T*>(sdata_);

   unsigned int tid = threadIdx.x;
   unsigned int i = blockIdx.x * 2 * blockSize + tid;
   unsigned int gridSize = blockSize * 2 * gridDim.x;
   sdata[tid] = 0.0f;
   while (i < n) 
   {
      sdata[tid] += g_idata[i];
      sdata[tid] += g_idata[i + blockSize]; 
      i += gridSize;
   }
   __syncthreads();
   if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
   if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
   if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
   if (blockSize >= 64) { if (tid < 32) { sdata[tid] += sdata[tid + 32]; } __syncthreads(); }
   if (blockSize >= 32) { if (tid < 16) { sdata[tid] += sdata[tid + 16]; } __syncthreads(); }
   if (blockSize >= 16) { if (tid < 8) { sdata[tid] += sdata[tid + 8]; } __syncthreads(); }
   if (blockSize >= 8) { if (tid < 4) { sdata[tid] += sdata[tid + 4]; } __syncthreads(); }
   if (blockSize >= 4) { if (tid < 2) { sdata[tid] += sdata[tid + 2]; } __syncthreads(); }
   if (blockSize >= 2) { if (tid < 1) { sdata[tid] += sdata[tid + 1]; } __syncthreads(); }
   if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; };
}



__device__ float3 warpPoint(int nD, const float3* D, float3 offset, int x, int y, int z)
{
   float Didx = (x * offset.x + y * offset.y + z * offset.z) * (nD-1);
   int idx = Didx; // => floor
   float f = Didx - idx;
   float3 p;
   p.x = f * D[idx + 1].x + (1 - f) * D[idx].x;
   p.y = f * D[idx + 1].y + (1 - f) * D[idx].y;
   p.z = f * D[idx + 1].z + (1 - f) * D[idx].z;
   return p;
}

__device__ float getPoint(int tex_id, float3 p)
{
   p.x += 0.5f;
   p.y += 0.5f;
   p.z += 0.5f;
   
   switch(tex_id)
   {
   case 0: return tex3D(tex0, p.x, p.y, p.z);
   case 1: return tex3D(tex1, p.x, p.y, p.z);
   case 2: return tex3D(tex2, p.x, p.y, p.z);
   case 3: return tex3D(tex3, p.x, p.y, p.z);
   case 4: return tex3D(tex4, p.x, p.y, p.z);
   case 5: return tex3D(tex5, p.x, p.y, p.z);
   case 6: return tex3D(tex6, p.x, p.y, p.z);
   case 7: return tex3D(tex7, p.x, p.y, p.z);
   case 8: return tex3D(tex8, p.x, p.y, p.z);
   case 9: return tex3D(tex9, p.x, p.y, p.z);
   case 10: return tex3D(tex10, p.x, p.y, p.z);
   case 11: return tex3D(tex11, p.x, p.y, p.z);
   case 12: return tex3D(tex12, p.x, p.y, p.z);
   case 13: return tex3D(tex13, p.x, p.y, p.z);
   case 14: return tex3D(tex14, p.x, p.y, p.z);
   case 15: return tex3D(tex15, p.x, p.y, p.z);   
   }  
   return 0;
}


__global__ void warp(int tex_id, int3 size, float3 offset, float* warp_img, int nD, float3* D)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;   
   int z = threadIdx.z + blockDim.z * blockIdx.z;
   int idx = x + y * size.x + z * (size.x * size.y);

   float3 p = warpPoint(nD, D, offset, x, y, z);
   p.x += x; p.y += y; p.z += z;
   float v = getPoint(tex_id, p);

   warp_img[idx] = (v != 0.0f) * (v - 1.0f);
}

__global__ void warpAndGetError(int tex_id, int3 size, float3 offset, float* __restrict__ reference, float* __restrict__ error_img, float* __restrict__ error_sq_img, int nD, float3* D)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;   
   int z = threadIdx.z + blockDim.z * blockIdx.z;
   int idx = x + y * size.x + z * (size.x * size.y);

   float3 p = warpPoint(nD, D, offset, x, y, z);
   p.x += x; p.y += y; p.z += z;
   float v = getPoint(tex_id, p);

   float mask = (v != 0.0f) ? 1.0f : 0.0f; // set out of range values to zero
   v -= (1.0f + reference[idx]); // we have added 1 to use zero as special case
   error_img[idx] = mask * v; 
   error_sq_img[idx] = mask * v * v; 
}

__global__ void warpIntensityPreserving(int tex_id, int3 size, float3 offset, float* warp_img, uint16_t* mask_img, int nD, float3* D)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;   
   int z = threadIdx.z + blockDim.z * blockIdx.z;

   float3 p0;
   p0.x = x; p0.y = y; p0.z = z;
   float v = getPoint(tex_id, p0);

   float3 p = warpPoint(nD, D, offset, x, y, z);
   p0 -= p;

   x = round(p0.x);
   y = round(p0.y);
   z = round(p0.z);
   
   uint16_t valid = (x >= 0) && (x < size.x) && (y >= 0) && (y < size.y) && (z >= 0) && (z < size.z); 
   int idx = x + y * size.x + z * (size.x * size.y);
   
   if (valid)
   {
      warp_img[idx] += (v != 0.0f) * (v - 1.0f);
      mask_img[idx]++;   
   }
}



__global__ void computeJacobian(float* error_img, float3* __restrict__ jac_, float3* __restrict__ VI_dW_dp_, int nD, int range_max, GpuRange* range)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   float3* jac = jac_ + x;
   float3* VI_dW_dp = VI_dW_dp_ + x;

   if (x < range_max)
      for (int i = 0; i < nD; i++)
      {
         int p0 = range[i].begin;
         int p1 = range[i].end;
         int p = x + p0;
         int idx = range_max*i;

         jac[idx].x = 0;
         jac[idx].y = 0;
         jac[idx].z = 0;   
         
         if (p <= p1)
         {
            float ep = error_img[p];
            jac[idx].x = VI_dW_dp[idx].x * ep;
            jac[idx].y = VI_dW_dp[idx].y * ep;
            jac[idx].z = VI_dW_dp[idx].z * ep;  
         }
      }
}


GpuFrame::GpuFrame(const cv::Mat& frame_)
{
   auto tex_manager = GpuTextureManager::instance();
   texture = tex_manager->getTextureId();
   
   auto& tex = getTexture(texture);
 
   frame = frame_;

   // Set texture parameters
   tex.addressMode[0] = hipAddressModeBorder;
   tex.addressMode[1] = hipAddressModeBorder;
   tex.addressMode[2] = hipAddressModeBorder;
   tex.filterMode = hipFilterModePoint;
   tex.normalized = false;

   // Add 1 to frame value -> we want to use zero as a special case
   cv::Mat frame_cpy;
   frame.copyTo(frame_cpy);
   frame_cpy += 1.0f;

   size.x = frame.size[2];
   size.y = frame.size[1];
   size.z = frame.size[0];

   // Allocate array and copy image data
   hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
   hipExtent extent = make_hipExtent(size.x, size.y, size.z);
   checkCudaErrors(hipMalloc3DArray(&cu_array, &channelDesc, extent));
   
   hipMemcpy3DParms copy_params = {0};
   copy_params.srcPtr = make_hipPitchedPtr((void*) frame_cpy.data, size.x * sizeof(float), size.x, size.y);
   copy_params.dstArray = cu_array;
   copy_params.extent = extent;
   copy_params.kind = hipMemcpyHostToDevice;
   checkCudaErrors(hipMemcpy3D(&copy_params));
   
   // Bind the array to the texture
   checkCudaErrors(hipBindTextureToArray(&tex, cu_array, &channelDesc));
}


GpuFrame::~GpuFrame()
{
   auto& tex = getTexture(texture);
   checkCudaErrors(hipUnbindTexture(tex));
   checkCudaErrors(hipFreeArray(cu_array));

   auto tex_manager = GpuTextureManager::instance();
   tex_manager->returnTextureId(texture);
}


GpuWorkingSpace::GpuWorkingSpace(int volume, int nD, int range_max, bool calculate_jacobian_on_gpu) : 
   calculate_jacobian_on_gpu(calculate_jacobian_on_gpu)
{
   checkCudaErrors(hipMalloc((void**) &error_sum, 1024 * sizeof(float)));
   checkCudaErrors(hipMalloc((void**) &error_image, volume*sizeof(float)));
   checkCudaErrors(hipMalloc((void**) &error_sq_image, volume*sizeof(float)));
   checkCudaErrors(hipMalloc((void**) &mask, volume*sizeof(uint16_t)));
   checkCudaErrors(hipMalloc((void**) &D, nD*sizeof(float3)));

   if (calculate_jacobian_on_gpu)   
      checkCudaErrors(hipMalloc((void**) &jacobian, range_max * nD * sizeof(float3)));

   hipStreamCreate(&stream);
}

GpuWorkingSpace::~GpuWorkingSpace()
{
   checkCudaErrors(hipFree(error_sum));
   checkCudaErrors(hipFree(error_image));
   checkCudaErrors(hipFree(mask));
   checkCudaErrors(hipFree(error_sq_image));
   checkCudaErrors(hipFree(D));

   if (calculate_jacobian_on_gpu)
      checkCudaErrors(hipFree(jacobian));

   hipStreamDestroy(stream);
}


GpuReferenceInformation::GpuReferenceInformation(const cv::Mat& ref_, float3 offset, int nD, int range_max, bool compute_jacobian_on_gpu) :
   offset(offset), nD(nD), range_max(range_max), compute_jacobian_on_gpu(compute_jacobian_on_gpu)
{
   size_t n_px = ref_.size[0] * ref_.size[1] * ref_.size[2];

   cvref = ref_;

   if (compute_jacobian_on_gpu)
   {
      checkCudaErrors(hipMalloc((void**) &VI_dW_dp, range_max * nD * sizeof(float3)));
      checkCudaErrors(hipMalloc((void**) &range, nD * sizeof(GpuRange)));   
   }

   checkCudaErrors(hipMalloc((void**) &reference, n_px * sizeof(float)));
   checkCudaErrors(hipMemcpy(reference, ref_.data, n_px * sizeof(float), hipMemcpyHostToDevice));
}

GpuReferenceInformation::~GpuReferenceInformation()
{
   if (compute_jacobian_on_gpu)
   {
      checkCudaErrors(hipFree(VI_dW_dp));
      checkCudaErrors(hipFree(range));   
   }
   checkCudaErrors(hipFree(reference));
}


void computeWarp(GpuFrame* frame, GpuWorkingSpace* w, GpuReferenceInformation* gpu_ref)
{
   auto size = frame->size;
   dim3 dimBlock(32, 32, 1);
   dim3 dimGrid(size.x / 32, size.y / 32, size.z);

   int id = frame->getTextureId();
   warp<<<dimGrid, dimBlock, 0, w->stream>>>(id, frame->size, gpu_ref->offset, w->error_image, gpu_ref->nD, w->D);
   getLastCudaError("Kernel execution failed [ warp ]");
}


void computeIntensityPreservingWarp(GpuFrame* frame, GpuWorkingSpace* w, GpuReferenceInformation* gpu_ref)
{
   auto size = frame->size;
   dim3 dimBlock(32, 32, 1);
   dim3 dimGrid(size.x / 32, size.y / 32, size.z);

   hipMemset(w->error_image, 0, size.x * size.y * size.z * sizeof(float));
   hipMemset(w->mask, 0, size.x * size.y * size.z * sizeof(uint16_t));
   
   int id = frame->getTextureId();
   warpIntensityPreserving<<<dimGrid, dimBlock, 0, w->stream>>>(id, frame->size, gpu_ref->offset, w->error_image, w->mask, gpu_ref->nD, w->D);
   getLastCudaError("Kernel execution failed [ warpIntensityPreserving ]");
}


double computeError(GpuFrame* frame, GpuWorkingSpace* w, GpuReferenceInformation* gpu_ref)
{
   auto size = frame->size;
   int volume = size.x * size.y * size.z;

   dim3 dimBlock(32, 32, 1);
   dim3 dimGrid(size.x / 32, size.y / 32, size.z);

   int id = frame->getTextureId();
   warpAndGetError<<<dimGrid, dimBlock, 0, w->stream>>>(id, frame->size, gpu_ref->offset, gpu_ref->reference, w->error_image, w->error_sq_image, gpu_ref->nD, w->D);
   getLastCudaError("Kernel execution failed [ reduceSum ]");

   const int block_size = 512;
   int n_block = 1; //volume / (block_size * 2);
   reduceSum<block_size><<<dim3(1,1,1), dim3(block_size,1,1), block_size * sizeof(float), w->stream>>> (w->error_sq_image, w->error_sum, volume);
   getLastCudaError("Kernel execution failed [ reduceSum ]");

   std::vector<float> error_sum(n_block);
   checkCudaErrors(hipMemcpy(error_sum.data(), w->error_sum, n_block*sizeof(float), hipMemcpyDeviceToHost));

   for(int i=1; i<n_block; i++)
      error_sum[0] += error_sum[i];

   return error_sum[0];
}

std::vector<float3> computeJacobianGpu(GpuFrame* frame, GpuWorkingSpace* w, GpuReferenceInformation* gpu_ref)
{  
   int range_max = gpu_ref->range_max;

   int block_size = 512;
   int n_block = (range_max + block_size - 1) / block_size;

   int id = frame->getTextureId();
   computeJacobian<<<n_block, block_size, 0, w->stream>>>(w->error_image, w->jacobian,
      gpu_ref->VI_dW_dp, gpu_ref->nD, range_max, gpu_ref->range);
   
   getLastCudaError("Kernel execution failed [ computeJacobian ]");

   std::vector<float3> jac_out(gpu_ref->nD);

   for(int i=0; i<gpu_ref->nD; i++)
   {
      int n_block = 1;
      const int block_size = 512;
      reduceSum<block_size><<<n_block, block_size, block_size * sizeof(float3), w->stream>>> ((f3*)w->jacobian + i * range_max, (f3*)w->error_sum, range_max);
      getLastCudaError("Kernel execution failed [ reduceSum ]");   
      
      std::vector<float3> jac_sum(n_block);
      checkCudaErrors(hipMemcpy(jac_sum.data(), w->error_sum, n_block*sizeof(float3), hipMemcpyDeviceToHost));   
      jac_out[i] = jac_sum[0];
   }   

   return jac_out;
}