#include "hip/hip_runtime.h"
#include "GpuFrameWarperKernels.h"
#include "GpuTextureManager.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"

texture<float, 3, hipReadModeElementType> tex0;
texture<float, 3, hipReadModeElementType> tex1;
texture<float, 3, hipReadModeElementType> tex2;
texture<float, 3, hipReadModeElementType> tex3;

float3 operator+(const float3& a, const float3& b)
{
   float3 c;
   c.x = a.x + b.x;
   c.y = a.y + b.y;
   c.z = a.z + b.z;
   return c;
}

__device__ float3& operator+=(float3& a, const float3& b)
{
   a.x += b.x;
   a.y += b.y;
   a.z += b.z;
   return a;
}

__device__ float3 operator-(const float3& a, const float3& b)
{
   float3 c;
   c.x = a.x - b.x;
   c.y = a.y - b.y;
   c.z = a.z - b.z;
   return c;
}

__device__ float3& operator-=(float3& a, const float3& b)
{
   a.x -= b.x;
   a.y -= b.y;
   a.z -= b.z;
   return a;
}

class f3
{
public:

   __device__ f3& operator=(float other)
   {
      x = other;
      y = other;
      z = other;
      return *this;
   }

   __device__ f3& operator=(f3& other)
   {
      x = other.x;
      y = other.y;
      z = other.z;
      return *this;
   }

   __device__ f3& operator+=(const f3& other)
   {
      x += other.x;
      y += other.y;
      z += other.z;
      return *this;
   }

   float x = 0;
   float y = 0;
   float z = 0;
};

GpuTextureManager* GpuTextureManager::gpu_texture_manager = nullptr;

texture<float, 3, hipReadModeElementType>& getTexture(int id)
{
   switch (id)
   {
   case 0: return tex0;
   case 1: return tex1;
   case 2: return tex2;
   case 3: return tex3;
   }
   throw std::runtime_error("Invalid texture reference");
}

GpuTextureManager* GpuTextureManager::instance()
{
   if (!gpu_texture_manager)
      gpu_texture_manager = new GpuTextureManager;
   return gpu_texture_manager;
}

int GpuTextureManager::getTextureId()
{
   std::unique_lock<std::mutex> lk(mutex);
   cv.wait(lk, [this] { return !free_textures.empty(); });
   int texture = free_textures.back();

   free_textures.pop_back();
   return texture;
}

void GpuTextureManager::returnTextureId(int t)
{
   {
      std::lock_guard<std::mutex> lk(mutex);
      free_textures.push_back(t);
   }
   cv.notify_one();
}

GpuTextureManager::GpuTextureManager()
{
   for (int i = 0; i < 4; i++)
      free_textures.push_back(i);
}


template <unsigned int blockSize, class T>
__device__ void warpReduce(T *sdata, unsigned int tid) {

}

template <unsigned int blockSize, class T>
__global__ void reduceSum(T *g_idata, T *g_odata, unsigned int n) {
   extern __shared__ char sdata_[];
   T* sdata = reinterpret_cast<T*>(sdata_);

   unsigned int tid = threadIdx.x;
   unsigned int i = blockIdx.x * 2 * blockSize + tid;
   unsigned int gridSize = blockSize * 2 * gridDim.x;
   sdata[tid] = 0.0f;
   while (i < n) 
   {
      sdata[tid] += g_idata[i];
      sdata[tid] += g_idata[i + blockSize]; 
      i += gridSize;
   }
   __syncthreads();
   if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
   if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
   if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
   if (blockSize >= 64) { if (tid < 32) { sdata[tid] += sdata[tid + 32]; } __syncthreads(); }
   if (blockSize >= 32) { if (tid < 16) { sdata[tid] += sdata[tid + 16]; } __syncthreads(); }
   if (blockSize >= 16) { if (tid < 8) { sdata[tid] += sdata[tid + 8]; } __syncthreads(); }
   if (blockSize >= 8) { if (tid < 4) { sdata[tid] += sdata[tid + 4]; } __syncthreads(); }
   if (blockSize >= 4) { if (tid < 2) { sdata[tid] += sdata[tid + 2]; } __syncthreads(); }
   if (blockSize >= 2) { if (tid < 1) { sdata[tid] += sdata[tid + 1]; } __syncthreads(); }
   if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; };
}



__device__ float3 warpPoint(int nD, const float3* D, float3 offset, int x, int y, int z)
{
   float Didx = (x * offset.x + y * offset.y + z * offset.z) * (nD-1);
   int idx = Didx; // => floor
   float f = Didx - idx;
   float3 p;
   p.x = f * D[idx + 1].x + (1 - f) * D[idx].x;
   p.y = f * D[idx + 1].y + (1 - f) * D[idx].y;
   p.z = f * D[idx + 1].z + (1 - f) * D[idx].z;
   return p;
}

__device__ float getPoint(int tex_id, float3 p)
{
   p.x += 0.5f;
   p.y += 0.5f;
   p.z += 0.5f;
   
   float v = 0;
   if (tex_id == 0)
      v = tex3D(tex0, p.x, p.y, p.z);
   else if (tex_id == 1)
      v = tex3D(tex1, p.x, p.y, p.z);
   else if (tex_id == 2)
      v = tex3D(tex2, p.x, p.y, p.z);
   else if (tex_id == 3)
      v = tex3D(tex3, p.x, p.y, p.z);
      
   return v;
}


__global__ void warp(int tex_id, int3 size, float3 offset, float* warp_img, int nD, float3* D)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;   
   int z = threadIdx.z + blockDim.z * blockIdx.z;
   int idx = x + y * size.x + z * (size.x * size.y);

   float3 p = warpPoint(nD, D, offset, x, y, z);
   p.x += x; p.y += y; p.z += z;
   float v = getPoint(tex_id, p);

   warp_img[idx] = (v != 0.0f) * (v - 1.0f);
}

__global__ void warpAndGetError(int tex_id, int3 size, float3 offset, float* __restrict__ reference, float* __restrict__ error_img, float* __restrict__ error_sq_img, int nD, float3* D)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;   
   int z = threadIdx.z + blockDim.z * blockIdx.z;
   int idx = x + y * size.x + z * (size.x * size.y);

   float3 p = warpPoint(nD, D, offset, x, y, z);
   p.x += x; p.y += y; p.z += z;
   float v = getPoint(tex_id, p);

   float mask = (v != 0.0f) ? 1.0f : 0.0f; // set out of range values to zero
   v -= (1.0f + reference[idx]); // we have added 1 to use zero as special case
   error_img[idx] = mask * v; 
   error_sq_img[idx] = mask * v * v; 
}

__global__ void warpIntensityPreserving(int tex_id, int3 size, float3 offset, float* warp_img, uint16_t* mask_img, int nD, float3* D)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;   
   int z = threadIdx.z + blockDim.z * blockIdx.z;

   float3 p0;
   p0.x = x; p0.y = y; p0.z = z;
   float v = getPoint(tex_id, p0);

   float3 p = warpPoint(nD, D, offset, x, y, z);
   p0 -= p;

   x = round(p0.x);
   y = round(p0.y);
   z = round(p0.z);
   
   uint16_t valid = (x >= 0) && (x < size.x) && (y >= 0) && (y < size.y) && (z >= 0) && (z < size.z); 
   int idx = x + y * size.x + z * (size.x * size.y);
   
   if (valid)
   {
      warp_img[idx] += (v != 0.0f) * (v - 1.0f);
      mask_img[idx]++;   
   }
}



__global__ void computeJacobian(float* error_img, float3* __restrict__ jac_, float3* __restrict__ VI_dW_dp_, int nD, int range_max, GpuRange* range)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   float3* jac = jac_ + x;
   float3* VI_dW_dp = VI_dW_dp_ + x;

   if (x < range_max)
      for (int i = 0; i < nD; i++)
      {
         int p0 = range[i].begin;
         int p1 = range[i].end;
         int p = x + p0;
         int idx = range_max*i;

         jac[idx].x = 0;
         jac[idx].y = 0;
         jac[idx].z = 0;   
         
         if (p <= p1)
         {
            float ep = error_img[p];
            jac[idx].x = VI_dW_dp[idx].x * ep;
            jac[idx].y = VI_dW_dp[idx].y * ep;
            jac[idx].z = VI_dW_dp[idx].z * ep;  
         }
      }
}


GpuFrame::GpuFrame(cv::Mat frame_, int nD)
{
   auto tex_manager = GpuTextureManager::instance();
   texture = tex_manager->getTextureId();
   
   auto& tex = getTexture(texture);
 
   frame = frame_;

   // Set texture parameters
   tex.addressMode[0] = hipAddressModeBorder;
   tex.addressMode[1] = hipAddressModeBorder;
   tex.addressMode[2] = hipAddressModeBorder;
   tex.filterMode = hipFilterModePoint;
   tex.normalized = false;

   // Add 1 to frame value -> we want to use zero as a special case
   cv::Mat frame_cpy;
   frame.copyTo(frame_cpy);
   frame_cpy += 1.0f;

   size.x = frame.size[2];
   size.y = frame.size[1];
   size.z = frame.size[0];
   size_t volume = size.x * size.y * size.z;

   // Allocate array and copy image data
   hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
   hipExtent extent = make_hipExtent(size.x, size.y, size.z);
   size_t copy_size = volume * sizeof(float);
   checkCudaErrors(hipMalloc3DArray(&cu_array, &channelDesc, extent));
   checkCudaErrors(hipMemcpyToArray(cu_array, 0, 0, frame_cpy.data, copy_size, hipMemcpyHostToDevice));

   // Bind the array to the texture
   checkCudaErrors(hipBindTextureToArray(&tex, cu_array, &channelDesc));

   checkCudaErrors(hipMalloc((void**) &error_sum, 1024 * sizeof(float)));
   checkCudaErrors(hipMalloc((void**) &error_image, volume*sizeof(float)));
   checkCudaErrors(hipMalloc((void**) &error_sq_image, volume*sizeof(float)));
   checkCudaErrors(hipMalloc((void**) &mask, volume*sizeof(uint16_t)));
   checkCudaErrors(hipMalloc((void**) &jacobian, 3*volume*sizeof(float3)));
   checkCudaErrors(hipMalloc((void**) &D, nD*sizeof(float3)));
}

GpuFrame::~GpuFrame()
{
   auto& tex = getTexture(texture);
   checkCudaErrors(hipUnbindTexture(tex));
   checkCudaErrors(hipFreeArray(cu_array));

   auto tex_manager = GpuTextureManager::instance();
   tex_manager->returnTextureId(texture);

   checkCudaErrors(hipFree(error_sum));
   checkCudaErrors(hipFree(error_image));
   checkCudaErrors(hipFree(mask));
   checkCudaErrors(hipFree(error_sq_image));
   checkCudaErrors(hipFree(jacobian));
   checkCudaErrors(hipFree(D));
}


GpuReferenceInformation::GpuReferenceInformation(const cv::Mat& ref_, float3 offset, int nD, int range_max) :
   offset(offset), nD(nD), range_max(range_max)
{
   size_t n_px = ref_.size[0] * ref_.size[1] * ref_.size[2];

   cvref = ref_;

   checkCudaErrors(hipMalloc((void**) &reference, n_px * sizeof(float)));
   checkCudaErrors(hipMalloc((void**) &VI_dW_dp, range_max * nD * sizeof(float3)));
   checkCudaErrors(hipMalloc((void**) &range, nD * sizeof(GpuRange)));

   checkCudaErrors(hipMemcpy(reference, ref_.data, n_px * sizeof(float), hipMemcpyHostToDevice));
}

GpuReferenceInformation::~GpuReferenceInformation()
{
   checkCudaErrors(hipFree(VI_dW_dp));
   checkCudaErrors(hipFree(range));
   checkCudaErrors(hipFree(reference));
}


void computeWarp(GpuFrame* frame, GpuReferenceInformation* gpu_ref)
{
   auto size = frame->size;
   dim3 dimBlock(32, 32, 1);
   dim3 dimGrid(size.x / 32, size.y / 32, size.z);

   int id = frame->getTextureId();
   warp<<<dimGrid, dimBlock, 0>>>(id, frame->size, gpu_ref->offset, frame->error_image, gpu_ref->nD, frame->D);
   getLastCudaError("Kernel execution failed [ warp ]");
}


void computeIntensityPreservingWarp(GpuFrame* frame, GpuReferenceInformation* gpu_ref)
{
   auto size = frame->size;
   dim3 dimBlock(32, 32, 1);
   dim3 dimGrid(size.x / 32, size.y / 32, size.z);

   hipMemset(frame->error_image, 0, size.x * size.y * size.z * sizeof(float));
   hipMemset(frame->mask, 0, size.x * size.y * size.z * sizeof(uint16_t));
   
   int id = frame->getTextureId();
   warpIntensityPreserving<<<dimGrid, dimBlock, 0>>>(id, frame->size, gpu_ref->offset, frame->error_image, frame->mask, gpu_ref->nD, frame->D);
   getLastCudaError("Kernel execution failed [ warpIntensityPreserving ]");
}


double computeError(GpuFrame* frame, GpuReferenceInformation* gpu_ref)
{
   auto size = frame->size;
   int volume = size.x * size.y * size.z;

   dim3 dimBlock(32, 32, 1);
   dim3 dimGrid(size.x / 32, size.y / 32, size.z);

   int id = frame->getTextureId();
   warpAndGetError<<<dimGrid, dimBlock, 0>>>(id, frame->size, gpu_ref->offset, gpu_ref->reference, frame->error_image, frame->error_sq_image, gpu_ref->nD, frame->D);
   getLastCudaError("Kernel execution failed [ reduceSum ]");

   const int block_size = 512;
   int n_block = 1; //volume / (block_size * 2);
   reduceSum<block_size><<<dim3(1,1,1), dim3(block_size,1,1), block_size * sizeof(float)>>> (frame->error_sq_image, frame->error_sum, volume);
   getLastCudaError("Kernel execution failed [ reduceSum ]");

   std::vector<float> error_sum(n_block);
   checkCudaErrors(hipMemcpy(error_sum.data(), frame->error_sum, n_block*sizeof(float), hipMemcpyDeviceToHost));

   for(int i=1; i<n_block; i++)
      error_sum[0] += error_sum[i];

   return error_sum[0];
}

std::vector<float3> computeJacobian(GpuFrame* frame, GpuReferenceInformation* gpu_ref)
{  
   int range_max = gpu_ref->range_max;

   int block_size = 512;
   int n_block = (range_max + block_size - 1) / block_size;

   int id = frame->getTextureId();
   computeJacobian<<<n_block, block_size, 0>>>(frame->error_image, frame->jacobian,
      gpu_ref->VI_dW_dp, gpu_ref->nD, range_max, gpu_ref->range);
   
   getLastCudaError("Kernel execution failed [ computeJacobian ]");

   std::vector<float3> jac_out(gpu_ref->nD);

   for(int i=0; i<gpu_ref->nD; i++)
   {
      int n_block = 1;
      const int block_size = 512;
      reduceSum<block_size><<<n_block, block_size, block_size * sizeof(float3)>>> ((f3*)frame->jacobian + i * range_max, (f3*)frame->error_sum, range_max);
      getLastCudaError("Kernel execution failed [ reduceSum ]");   
      
      std::vector<float3> jac_sum(n_block);
      checkCudaErrors(hipMemcpy(jac_sum.data(), frame->error_sum, n_block*sizeof(float3), hipMemcpyDeviceToHost));   
      jac_out[i] = jac_sum[0];
   }   

   return jac_out;
}