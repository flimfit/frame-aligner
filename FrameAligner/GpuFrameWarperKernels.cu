#include "hip/hip_runtime.h"
#include "GpuFrameWarperKernels.h"
#include "GpuTextureManager.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"

texture<float, 3, hipReadModeElementType> tex0;
texture<float, 3, hipReadModeElementType> tex1;
texture<float, 3, hipReadModeElementType> tex2;
texture<float, 3, hipReadModeElementType> tex3;
texture<float, 3, hipReadModeElementType> tex4;
texture<float, 3, hipReadModeElementType> tex5;
texture<float, 3, hipReadModeElementType> tex6;
texture<float, 3, hipReadModeElementType> tex7;
texture<float, 3, hipReadModeElementType> tex8;
texture<float, 3, hipReadModeElementType> tex9;
texture<float, 3, hipReadModeElementType> tex10;
texture<float, 3, hipReadModeElementType> tex11;
texture<float, 3, hipReadModeElementType> tex12;
texture<float, 3, hipReadModeElementType> tex13;
texture<float, 3, hipReadModeElementType> tex14;
texture<float, 3, hipReadModeElementType> tex15;


texture<float, 3, hipReadModeElementType>& getTexture(int id)
{
   switch (id)
   {
   case 0: return tex0;
   case 1: return tex1;
   case 2: return tex2;
   case 3: return tex3;
   case 4: return tex4;
   case 5: return tex5;
   case 6: return tex6;
   case 7: return tex7;
   case 8: return tex8;
   case 9: return tex9;
   case 10: return tex10;
   case 11: return tex11;
   case 12: return tex12;
   case 13: return tex13;
   case 14: return tex14;
   case 15: return tex15;
   }
   throw std::runtime_error("Invalid texture reference");
}

GpuTextureManager* GpuTextureManager::instance()
{
   if (!gpu_texture_manager)
      gpu_texture_manager = new GpuTextureManager;
   return gpu_texture_manager;
}

int GpuTextureManager::getTextureId()
{
   std::unique_lock<std::mutex> lk(mutex);
   cv.wait(lk, [this] { return !free_textures.empty(); });
   int texture = free_textures.back();

   free_textures.pop_back();
   return texture;
}

void GpuTextureManager::returnTextureId(int t)
{
   {
      std::lock_guard<std::mutex> lk(mutex);
      free_textures.push_back(t);
   }
   cv.notify_one();
}

GpuTextureManager::GpuTextureManager()
{
   for (int i = 0; i < 16; i++)
      free_textures.push_back(i);
}



template <unsigned int blockSize>
__device__ void warpReduce(float *sdata, unsigned int tid) {
   if (blockSize >= 64) { sdata[tid] += sdata[tid + 32]; }
   if (blockSize >= 32) { sdata[tid] += sdata[tid + 16]; }
   if (blockSize >= 16) { sdata[tid] += sdata[tid + 8]; }
   if (blockSize >= 8) { sdata[tid] += sdata[tid + 4]; }
   if (blockSize >= 4) { sdata[tid] += sdata[tid + 2]; }
   if (blockSize >= 2) { sdata[tid] += sdata[tid + 1]; }
}

template <unsigned int blockSize>
__global__ void reduceSum(float *g_idata, float *g_odata, unsigned int n) {
   extern __shared__ float sdata[];
   unsigned int tid = threadIdx.x;
   unsigned int i = blockIdx.x*(blockSize * 2) + tid;
   unsigned int gridSize = blockSize * 2 * gridDim.x;
   sdata[tid]= 0;
   while (i < n) 
   {
      sdata[tid] += g_idata[i] + g_idata[i + blockSize]; 
      i += gridSize;
   }
   __syncthreads();
   if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
   if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
   if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
   if (tid < 32) warpReduce<blockSize>(sdata, tid);
   if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; };
}



__device__ float3 warpPoint(int nD, float3* D, float3 offset, int x, int y, int z)
{
   float Didx = (x * offset.x + y * offset.y + z * offset.z) * (nD-1);
   int idx = Didx; // => floor
   float f = Didx - idx;
   float3 p;
   p.x = f * D[idx + 1].x + (1 - f) * D[idx].x;
   p.y = f * D[idx + 1].y + (1 - f) * D[idx].y;
   p.z = f * D[idx + 1].z + (1 - f) * D[idx].z;
   return p;
}


__global__ void warpAndGetError(int3 size, float3 offset, float* reference, float* error_img, int nD, float3* D)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;   
   int z = threadIdx.z + blockDim.z * blockIdx.z;
   int idx = x + y * size.x + z * (size.x * size.y);

   float3 p = warpPoint(nD, D, offset, x, y, z);

   float v = tex3D(tex0, p.x, p.y, p.z);
   float censor = v > 0; // set out of range values to zero

   // we have added 1 to use zero as special case
   error_img[idx] = censor * (v - 1.0 - reference[idx]); 
}

/*

__global__ void computeJacobian(float* error_img, float* jac, int nD, int n_dim)
{
   int p = threadIdx.x + blockDim.x * blockIdx.x;

   for (int i = 1; i < nD; i++)
   {
      int p0 = D_range[i - 1].begin;
      int p1 = D_range[i - 1].end;
      for (int p = p0; p < p1; p++)
      {
         jac[i*n_dim] += VI_dW_dp_x[i][p] * err_ptr[p]; // x 
         jac[i*n_dim + 1] += VI_dW_dp_y[i][p] * err_ptr[p]; // y
         if (n_dim == 3)
            jac(i*n_dim + 2) += VI_dW_dp_z[i][p] * err_ptr[p]; // z        
      }
   }
   for (int i = 0; i < (nD - 1); i++)
   {
      int p0 = D_range[i].begin;
      int p1 = D_range[i].end;
      for (int p = p0; p < p1; p++)
      {
         jac(i*n_dim) += VI_dW_dp_x[i][p] * err_ptr[p]; // x
         jac(i*n_dim + 1) += VI_dW_dp_y[i][p] * err_ptr[p]; // y
         if (n_dim == 3)
            jac(i*n_dim + 2) += VI_dW_dp_z[i][p] * err_ptr[p]; // z
         
      }
   }
   
}
*/




GpuFrame::GpuFrame(cv::Mat frame)
{
   auto tex_manager = GpuTextureManager::instance();
   texture = tex_manager->getTextureId();
   
   auto& tex = getTexture(texture);
  
   // Set texture parameters
   tex.addressMode[0] = hipAddressModeBorder;
   tex.addressMode[1] = hipAddressModeBorder;
   tex.filterMode = hipFilterModeLinear;
   tex.normalized = false;

   // Add 1 to frame value -> we want to use zero as a special case
   cv::Mat frame_cpy;
   frame.copyTo(frame_cpy);
   frame_cpy += 1.0f;

   // Allocate array and copy image data
   hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
   hipExtent extent = make_hipExtent(frame.size[0], frame.size[1], frame.size[2]);
   size_t copy_size = frame.size[0] * frame.size[1] * frame.size[2] * sizeof(float);
   checkCudaErrors(hipMalloc3DArray(&cu_array, &channelDesc, extent));
   checkCudaErrors(hipMemcpyToArray(cu_array, 0, 0, frame.data, copy_size, hipMemcpyHostToDevice));

   // Bind the array to the texture
   checkCudaErrors(hipBindTextureToArray(tex, cu_array, channelDesc));
}

GpuFrame::~GpuFrame()
{
   checkCudaErrors(hipFree(&cu_array));

   auto tex_manager = GpuTextureManager::instance();
   texture = tex_manager->getTextureId();
   auto& tex = getTexture(texture);
   checkCudaErrors(hipUnbindTexture(tex));
}